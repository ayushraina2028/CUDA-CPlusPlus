
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

#define threadsPerBlock 16

using namespace std;
using namespace chrono;

/*
Interleaved Addressing - Optimization 1
1. Replace divergent branch in inner loop.
2. With strided index and non divergent branch.

This introduces a new problem: Shared Memory Bank Conflicts.
*/

__global__ void Reduce1(int* device_input, int* device_output, int N) {
    
    __shared__ int s_data[threadsPerBlock];
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    s_data[tid] = (gid < N) ? device_input[gid] : 0;
    __syncthreads();

    for(int stride = 1; stride < blockDim.x; stride *= 2) {
        
        int index = 2 * stride * tid;
        if(index < blockDim.x) {
            s_data[index] += s_data[index + stride];
        }
        __syncthreads();

    }

    if(tid == 0) {
        device_output[blockIdx.x] = s_data[0];
    }
}

high_resolution_clock::time_point getTime() {
    return high_resolution_clock::now();
}   

int main() {
    int N = 1 << 28;
    size_t bytes = N*sizeof(int);

    int* host_input = new int[N];
    int* host_output = (int*) malloc((N/threadsPerBlock)*sizeof(int));

    // Initialize the input array to 1
    for(int i = 0;i < N; i++) {
        host_input[i] = 1;
    }

    // Pointers for GPU
    int* device_input; hipMalloc((void**)&device_input,bytes); hipMemcpy(device_input, host_input, bytes, hipMemcpyHostToDevice);
    int* device_output; hipMalloc((void**)&device_output,(N/threadsPerBlock)*sizeof(int));

    // Each thread Handles 2 Elements
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    cout << "Blocks Per Grid: " << blocksPerGrid << endl; cout << "Threads Per Block: " << threadsPerBlock << endl;

    // Kernel Call to Reduce the Array

    auto start = getTime();

    Reduce1<<<blocksPerGrid,threadsPerBlock>>>(device_input,device_output,N);
    hipDeviceSynchronize();

    while(blocksPerGrid > 1) {

        int newBlocksPerGrid = (blocksPerGrid + threadsPerBlock - 1) / threadsPerBlock;
        cout << "Entered While Loop!, Next Blocks Per Grid: " << newBlocksPerGrid << endl;
        
        Reduce1<<<newBlocksPerGrid,threadsPerBlock>>>(device_output, device_output, N);
        hipDeviceSynchronize();

        blocksPerGrid = newBlocksPerGrid;
    }

    auto stop = getTime();
    milliseconds duration = duration_cast<milliseconds>(stop - start);
    cout << "Time Taken: " << duration.count() << " ms" << endl;

    hipMemcpy(host_output,device_output,(N/threadsPerBlock)*sizeof(int),hipMemcpyDeviceToHost);
    hipFree(device_input); hipFree(device_output);

    cout << "Array Sum: " << host_output[0] << endl;
    int sumCPU = 0; for(int i = 0;i < N; i++) sumCPU += host_input[i]; 
    
    if(sumCPU == host_output[0]) cout << "CPU and GPU Sums Match!" << endl;
    else cout << "CPU and GPU Sums Do Not Match!" << endl;

    // cout << "Host Output Array: "; for(int i = 0;i < N/threadsPerBlock; i++) cout << host_output[i] << " "; cout << endl;

    delete[] host_input; free(host_output);
    return 0;
}