
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void addIntegers(int* a, int* b, int* c) {
    *c = *a + *b;
}

int main() {
    
    int a, b, c;
    int *dA, *dB, *dC;

    size_t size = sizeof(int);

    // Allocate space on GPU
    hipMalloc(&dA,size);
    hipMalloc(&dB,size);
    hipMalloc(&dC,size);

    // Initialize;
    a = 4;
    b = 4;

    // Copy
    hipMemcpy(dA,&a,size,hipMemcpyHostToDevice);
    hipMemcpy(dB,&b,size,hipMemcpyHostToDevice);

    // Launch kernel
    addIntegers<<<1,1>>> (dA,dB,dC);

    // Copy back to CPU
    hipMemcpy(&c,dC,size,hipMemcpyDeviceToHost);

    // Free memory
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    cout << "Answer is -> " << c << endl;
    return 0;
}