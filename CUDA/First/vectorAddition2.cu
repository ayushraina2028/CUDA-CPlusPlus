
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

__global__ void VectorAdditionUsingParallelThreads(int* A, int* B, int* C) {
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main() {
    int N = 512;
    size_t size = N*sizeof(int);

    int* a = (int*) malloc(size);
    int* b = (int*) malloc(size);
    int* c = (int*) malloc(size);

    int* dA;
    hipMalloc((void**)&dA,size);

    int* dB;
    hipMalloc((void**)&dB,size);

    int* dC;
    hipMalloc((void**)&dC,size);

    for(int i = 0;i < N; i++) {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }

    hipMemcpy(dA,a,size,hipMemcpyHostToDevice);
    hipMemcpy(dB,b,size,hipMemcpyHostToDevice);

    int threadsPerBlock = N;
    int numBlocks = 1;

    VectorAdditionUsingParallelThreads<<<numBlocks,threadsPerBlock>>> (dA,dB,dC);

    hipMemcpy(c,dC,size,hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    cout << "vector addition using parallel threads" << endl;
    for(int i = 0;i < 10; i++) {
        cout << a[i] << " + " << b[i] << " = " << c[i] << endl;
    }

    free(a);
    free(b);
    free(c);

    return 0;
}